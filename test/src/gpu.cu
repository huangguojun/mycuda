#include "hip/hip_runtime.h"
#include <iostream>
#include "gpu.hpp"

void printCudaVersion()
{
    std::cout << "CUDA Compiled version: " << __CUDACC_VER__ << std::endl;

    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}



__device__ int addem( int a, int b ) 
{
    return a + b;
}

__global__ void add( int a, int b, int *c ) 
{
    *c = addem( a, b );
}

void SimpleDeviceCall()
{

    int c;
    int *dev_c;
    hipMalloc( (void**)&dev_c, sizeof(int) );

    add<<<1,1>>>( 2, 7, dev_c );

    hipMemcpy( &c, dev_c, sizeof(int),hipMemcpyDeviceToHost );
    printf( "2 + 7 = %d\n", c );

    hipFree( dev_c ) ;
}


