#include "hip/hip_runtime.h"
#include <iostream>
#include "gpu.hpp"

void printCudaVersion()
{
    std::cout << "CUDA Compiled version: " << __CUDACC_VER_MAJOR__ << "." <<
        __CUDACC_VER_MINOR__ << "." <<__CUDACC_VER_BUILD__ << std::endl;

    int runtime_ver;
    hipRuntimeGetVersion(&runtime_ver);
    std::cout << "CUDA Runtime version: " << runtime_ver << std::endl;

    int driver_ver;
    hipDriverGetVersion(&driver_ver);
    std::cout << "CUDA Driver version: " << driver_ver << std::endl;
}



__device__ int addem( int a, int b ) 
{
    return a + b;
}

__global__ void add( int a, int b, int *c ) 
{
    *c = addem( a, b );
}

void SimpleDeviceCall()
{

    int c;
    int *dev_c;
    hipMalloc( (void**)&dev_c, sizeof(int) );

    add<<<1,1>>>( 2, 7, dev_c );

    hipMemcpy( &c, dev_c, sizeof(int),hipMemcpyDeviceToHost );
    printf( "2 + 7 = %d\n", c );

    hipFree( dev_c ) ;
}


void EnumGPU()
{
    hipDeviceProp_t  prop;

    int count;
    hipGetDeviceCount( &count );
    printf( "   --- device count %d ---\n", count );
    
    for (int i=0; i< count; i++) 
    {
        hipGetDeviceProperties( &prop, i ) ;
        printf( "   --- General Information for device %d ---\n", i );
        printf( "Name:  %s\n", prop.name );
        printf( "Compute capability:  %d.%d\n", prop.major, prop.minor );
        printf( "Clock rate:  %d\n", prop.clockRate );
        printf( "Device copy overlap:  " );
        if (prop.deviceOverlap)
          printf( "Enabled\n" );
        else
          printf( "Disabled\n");
        printf( "Kernel execution timeout :  " );
        if (prop.kernelExecTimeoutEnabled)
          printf( "Enabled\n" );
        else
          printf( "Disabled\n" );

        printf( "   --- Memory Information for device %d ---\n", i );
        printf( "Total global mem:  %ld\n", prop.totalGlobalMem );
        printf( "Total constant Mem:  %ld\n", prop.totalConstMem );
        printf( "Max mem pitch:  %ld\n", prop.memPitch );
        printf( "Texture Alignment:  %ld\n", prop.textureAlignment );

        printf( "   --- MP Information for device %d ---\n", i );
        printf( "Multiprocessor count:  %d\n",
                    prop.multiProcessorCount );
        printf( "Shared mem per mp:  %ld\n", prop.sharedMemPerBlock );
        printf( "Registers per mp:  %d\n", prop.regsPerBlock );
        printf( "Threads in warp:  %d\n", prop.warpSize );
        printf( "Max threads per block:  %d\n",
                    prop.maxThreadsPerBlock );
        printf( "Max thread dimensions:  (%d, %d, %d)\n",
                    prop.maxThreadsDim[0], prop.maxThreadsDim[1],
                    prop.maxThreadsDim[2] );
        printf( "Max grid dimensions:  (%d, %d, %d)\n",
                    prop.maxGridSize[0], prop.maxGridSize[1],
                    prop.maxGridSize[2] );
        printf( "\n" );
    }
}
